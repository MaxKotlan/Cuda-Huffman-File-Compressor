#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "FileContents.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char** argv){
    FileContents f = LoadFile("lotr.txt");
    //printFileContents(f);
    printf("\nFile is %ld bytes long\n", f.length);

    int hashmap[256];
    for (int i = 0; i < 256; i++)
        hashmap[i] = 0;
    for (int i = 0; i < f.length; i++){
        hashmap[(int)f.buffer[i]]++;
    }

    int bytesUsed = 0;
    int columnwidth = 15;
    for (int i = 0; i < 256; i++){
        if (hashmap[i] != 0){
            bytesUsed++;
            printf("%c: %d, ", i, hashmap[i]);
            if (bytesUsed%10 == 0) printf("\n");
        }
    }

    printf("\n\nElements in Hashmap Used: %d / 256\n", bytesUsed);
}