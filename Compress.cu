#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "FileContents.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct CharacterFrequency{
    char character = 0;
    int frequency = rand()%3000;
};

void swap(CharacterFrequency& lhs, CharacterFrequency& rhs){
    CharacterFrequency temp = lhs;
    lhs = rhs;
    rhs = temp;
}

void print(CharacterFrequency cf){
    printf("\n\tcharacter: %c\nfrequency: %d\n\n", cf.character, cf.frequency);
}

int cmp(const void *a, const void *b){
    return ((CharacterFrequency *)b)->frequency - ((CharacterFrequency *)a)->frequency;
}

void getBinary(int n)
{
	int loop;
	/*loop=15 , for 16 bits value, 15th bit to 0th bit*/
    bool preceding = false;
    for(loop=15; loop>=0; loop--)
	{
		if( (1 << loop) & n){
            preceding = true;
            printf("1");
        }else if(preceding || (n==0 && loop ==0)){
            //if (preceding)
            printf("0");
        }
	}
}

int getBitCount(int n){
	/*loop=15 , for 16 bits value, 15th bit to 0th bit*/
    bool preceding = false;
    int count = 0;
    for(int loop=15; loop>=0; loop--)
	{
		if( (1 << loop) & n){
            preceding = true;
            count++;
            //printf("1");
        }else if(preceding || (n==0 && loop ==0)){
            //if (preceding)
            count++;
            //printf("0");
        }
    }
    return count;
}

int main(int argc, char** argv){
    FileContents f = LoadFile("lotr.txt");
    //printFileContents(f);
    printf("\nFile is %ld bytes long\n", f.length);

    //int hashmap[256];
    const int hashmapsize = 256;
    CharacterFrequency hashmap[hashmapsize];

    for (int i = 0; i < f.length; i++){
        unsigned char letter = f.buffer[i];
        hashmap[letter].character=letter;
        hashmap[letter].frequency++;
    }

    /*
    int globalIndex = 128;
    for (int i = 64+1; i > 0; i/=2){
        for (int k = 0; k < i; k++){
            int parentIndex = k+globalIndex;
            int leftChildIndex = parentIndex / 2 -1;
            int rightChildIndex = parentIndex / 2 -2;
            printf("[ Parent: %d lc: %d rc: %d ]", parentIndex, leftChildIndex,rightChildIndex);
        }
        printf("\n");
        globalIndex+=i;
    }*/

    /*
    for (int i = 0; i < 128; i++){
        int parentIndex = i;
        int leftChildIndex = parentIndex * 2 +1;
        int rightChildIndex = parentIndex * 2 +2;
        //printf("[ Parent: %d lc: %d rc: %d ]", parentIndex, leftChildIndex,rightChildIndex);
    }*/

    /*Convert Hashmap into a Heap Using Bottom Up Heap Construction*/
    for (int i = hashmapsize/2 -1; i >= 0; i--){
        int parentIndex = i;
        int leftChildIndex = parentIndex * 2 +1;
        int rightChildIndex = parentIndex * 2 +2;
        

        int compareIndex = leftChildIndex;
        if (rightChildIndex < hashmapsize && hashmap[rightChildIndex].frequency < hashmap[leftChildIndex].frequency)
            compareIndex = rightChildIndex;

        if (hashmap[parentIndex].frequency > hashmap[compareIndex].frequency)
            swap(hashmap[parentIndex], hashmap[compareIndex]);
    }

    for (int i = 0; i < hashmapsize; i++){
        printf("%d ", hashmap[i].frequency);
    }


    //qsort(hashmap, 256, sizeof(CharacterFrequency), cmp);

    //int frequencyTree[256*2 - 1];

    //int freqencyIndex[256];

    //for (int i = 0; i < 256; i++){
        //freqencyIndex[hashmap[i].character] = 0;
    //    if (hashmap[i].frequency > 0){
            //getBinary(i);
    //       freqencyIndex[hashmap[i].character] = i;
            //printf(" ");
            //printf(" %c %d \n", hashmap[i].character, hashmap[i].frequency);
    //    }
    //}

    //int compressedBits = 0;
    //for (int i = 0; i < f.length; i++){
    //    unsigned char letter = f.buffer[i];
    //    compressedBits += getBitCount(freqencyIndex[letter]);
        //printf(" ");
    //}

    //printf("Compressed Bits: %d. Compressed Bytes: %d. Perentage: %f\n", compressedBits, compressedBits/8+1, (float)(compressedBits/8+1) / (float)f.length);

    //int frequencyTree[256*2 - 1];
    //for ()

    /*
    int bytesUsed = 0;
    int columnwidth = 15;
    for (int i = 0; i < 256; i++){
        if (hashmap[i] != 0){
            bytesUsed++;
            printf("%c: %d, ", i, hashmap[i]);
            if (bytesUsed%10 == 0) printf("\n");
        }
    }*/

    //printf("\n\nElements in Hashmap Used: %d / 256\n", bytesUsed);
}