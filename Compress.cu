#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "FileContents.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct CharacterFrequency{
    char character = 0;
    int frequency = 0;
};

int cmp(const void *a, const void *b){
    return ((CharacterFrequency *)b)->frequency - ((CharacterFrequency *)a)->frequency;
}

void getBinary(int n)
{
	int loop;
	/*loop=15 , for 16 bits value, 15th bit to 0th bit*/
    bool preceding = false;
    for(loop=15; loop>=0; loop--)
	{
		if( (1 << loop) & n){
            preceding = true;
            printf("1");
        }else if(preceding || (n==0 && loop ==0)){
            //if (preceding)
            printf("0");
        }
	}
}

int getBitCount(int n){
	/*loop=15 , for 16 bits value, 15th bit to 0th bit*/
    bool preceding = false;
    int count = 0;
    for(int loop=15; loop>=0; loop--)
	{
		if( (1 << loop) & n){
            preceding = true;
            count++;
            //printf("1");
        }else if(preceding || (n==0 && loop ==0)){
            //if (preceding)
            count++;
            //printf("0");
        }
    }
    return count;
}

int main(int argc, char** argv){
    FileContents f = LoadFile("lotr.txt");
    //printFileContents(f);
    printf("\nFile is %ld bytes long\n", f.length);

    //int hashmap[256];
    CharacterFrequency hashmap[256];

    for (int i = 0; i < f.length; i++){
        unsigned char letter = f.buffer[i];
        hashmap[letter].character=letter;
        hashmap[letter].frequency++;
    }
    qsort(hashmap, 256, sizeof(CharacterFrequency), cmp);

    //int frequencyTree[256*2 - 1];

    int freqencyIndex[256];

    for (int i = 0; i < 256; i++){
        //freqencyIndex[hashmap[i].character] = 0;
        if (hashmap[i].frequency > 0){
            //getBinary(i);
            freqencyIndex[hashmap[i].character] = i;
            //printf(" ");
            //printf(" %c %d \n", hashmap[i].character, hashmap[i].frequency);
        }
    }

    int compressedBits = 0;
    for (int i = 0; i < f.length; i++){
        unsigned char letter = f.buffer[i];
        compressedBits += getBitCount(freqencyIndex[letter]);
        //printf(" ");
    }

    printf("Compressed Bits: %d. Compressed Bytes: %d. Perentage: %f\n", compressedBits, compressedBits/8+1, (float)(compressedBits/8+1) / (float)f.length);

    //int frequencyTree[256*2 - 1];
    //for ()

    /*
    int bytesUsed = 0;
    int columnwidth = 15;
    for (int i = 0; i < 256; i++){
        if (hashmap[i] != 0){
            bytesUsed++;
            printf("%c: %d, ", i, hashmap[i]);
            if (bytesUsed%10 == 0) printf("\n");
        }
    }*/

    //printf("\n\nElements in Hashmap Used: %d / 256\n", bytesUsed);
}