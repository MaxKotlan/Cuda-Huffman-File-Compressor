#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "FileContents.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct CharacterFrequency{
    char character = 0;
    int frequency = 0;
};

int cmp(const void *a, const void *b){
    return ((CharacterFrequency *)b)->frequency - ((CharacterFrequency *)a)->frequency;
}

int main(int argc, char** argv){
    FileContents f = LoadFile("lotr.txt");
    //printFileContents(f);
    printf("\nFile is %ld bytes long\n", f.length);

    //int hashmap[256];
    CharacterFrequency hashmap[256];



    for (int i = 0; i < f.length; i++){
        char letter = f.buffer[i];
        hashmap[letter].character=letter;
        hashmap[letter].frequency++;
    }


    qsort(hashmap, 256, sizeof(CharacterFrequency), cmp);

    for (int i = 0; i < 256; i++){
        if (hashmap[i].frequency > 0){
            printf(" %c %d \n", hashmap[i].character, hashmap[i].frequency);
        }
    }

    /*
    int bytesUsed = 0;
    int columnwidth = 15;
    for (int i = 0; i < 256; i++){
        if (hashmap[i] != 0){
            bytesUsed++;
            printf("%c: %d, ", i, hashmap[i]);
            if (bytesUsed%10 == 0) printf("\n");
        }
    }*/

    //printf("\n\nElements in Hashmap Used: %d / 256\n", bytesUsed);
}