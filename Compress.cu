#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "FileContents.h"
#include "CharacterFrequency.h"
#include "Heap.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char** argv){
    FileContents f = LoadFile("lotr.txt");
    //printFileContents(f);
    printf("\nFile is %ld bytes long\n", f.length);

    /*Using a Hashmap, find the frequency of each character in the file*/
    const int hashmapsize = 256;
    CharacterFrequency hashmap[hashmapsize];

    for (int i = 0; i < f.length; i++){
        unsigned char letter = f.buffer[i];
        hashmap[letter].character=letter;
        hashmap[letter].frequency++;
    }

    /*Convert Hashmap into a Heap Using Bottom Up Heap Construction*/
    for (int i = hashmapsize/2 -1; i >= 0; i--){
        int parentIndex = i;
        int leftChildIndex = parentIndex * 2 +1;
        int rightChildIndex = parentIndex * 2 +2;
        
        int compareIndex = leftChildIndex;
        if (rightChildIndex < hashmapsize && hashmap[rightChildIndex].frequency < hashmap[leftChildIndex].frequency)
            compareIndex = rightChildIndex;

        if (hashmap[parentIndex].frequency > hashmap[compareIndex].frequency)
            swap(hashmap[parentIndex], hashmap[compareIndex]);
    }

    FrequencyHeap fh = {hashmap, hashmapsize};
    
    CharacterFrequency freq = RemoveMinimumFrequencyFromHeap(fh);
    while (fh.lastElementIndex > 0) {
        //printf("%d ", freq.frequency);
        RemoveMinimumFrequencyFromHeap(fh);
    };
    //RemoveMinimumFrequencyFromHeap(fh);
    //printf("Removed %d \n", RemoveMinimumFrequencyFromHeap(fh).frequency);

    //
    //for (int i = 0; i < fh.lastElementIndex; i++){
    //    printf("%d ", fh.elements[i].frequency);
    //}
}